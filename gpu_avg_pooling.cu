#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<algorithm>
#include<time.h>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void avg_pooling(float* dev, float* gpu_output_data, int input_h_size, int input_w_size, int pool_h_size, int pool_w_size, int pool_h_stride, int pool_w_stride)
{
        int x = blockIdx.x;
	int y = blockIdx.y;

	int sum;
        float avg;

        int pooled_size = ((input_w_size - pool_w_size) / pool_w_stride) + 1;
	int h_start = y * pool_h_stride;
        int w_start = x * pool_w_stride;
        int h_end = min(h_start + pool_h_size, input_h_size);
        int w_end = min(w_start + pool_w_size, input_w_size);

        h_start = max(h_start, 0);
        w_start = max(w_start, 0);
        sum = 0;
        avg = 0;

        int pool_index = (y * pooled_size) + x;
        for (int h = h_start; h < h_end; h++)
        {
            for (int w = w_start; w < w_end; w++)
            {
                  	int index = (h * input_w_size) + w;
                   	sum += dev[index];
            }
        avg = (float)sum / (pool_h_size * pool_w_size);
        gpu_output_data[pool_index] = avg;
        }
}
void Init_input(float* input, int input_h_size, int input_w_size, int num)
{
        srand(time(NULL));

        for (int h = 0; h < input_h_size; h++)
        {
        	for (int w = 0; w < input_w_size; w++)
                {
                	input[(h * input_w_size) + w] = rand() % num;
                }
        }

}
void print(float* data, int h_size, int w_size)
{
	for (int h = 0; h < h_size; h++)
        {
                for (int w = 0; w < w_size; w++)
                {
           	        printf("%.2f ", data[(h * w_size) + w]);
		}
	        printf("\n");
        }
        printf("\n");
}
int main()
{
	int input_h_size = 100;
	int input_w_size = 100;
	int pool_w_size = 99;
        int pool_h_size = 99;
        int pool_w_stride = 1;
        int pool_h_stride = 1;
	
	int pooled_h = ((input_h_size - pool_h_size) / pool_h_stride) + 1;
        int pooled_w = ((input_w_size - pool_w_size) / pool_w_stride) + 1;	

	float* input = (float*)malloc(sizeof(float) * input_h_size * input_w_size);
	float* result = (float*)malloc(sizeof(float) * input_h_size * input_w_size);
	float* cpu_result = (float*)malloc(sizeof(float) * input_h_size * input_w_size);
	float* gpu_output_data;
	float* dev;

	Init_input(input, input_h_size, input_w_size, 10);

	print(input, input_h_size, input_w_size);

	hipMalloc((void**)&dev, sizeof(float) * input_h_size * input_w_size);
	hipMalloc((void**)&gpu_output_data, sizeof(float) * input_h_size * input_w_size);

	hipMemcpy(dev, input, sizeof(float) * input_h_size * input_w_size, hipMemcpyHostToDevice);
	
	dim3 dimGrid(pooled_h, pooled_w);
	avg_pooling<<<dimGrid,1>>>(dev, gpu_output_data, input_h_size, input_w_size, pool_h_size, pool_w_size, pool_h_stride, pool_w_stride);
	
	hipMemcpy(result, gpu_output_data, sizeof(float) * input_h_size * input_w_size, hipMemcpyDeviceToHost);

	print(result, pooled_h, pooled_w);
	
	hipFree(gpu_output_data);
	hipFree(dev);
	free(input);
	free(result);

	return 0; 
}



































